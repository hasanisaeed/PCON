#include "hip/hip_runtime.h"
__global__ void computeKernel(float* data) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Perform computation
    data[idx] = /* computation logic */;
}
